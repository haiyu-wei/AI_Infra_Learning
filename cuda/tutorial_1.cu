
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello from gpu");
}

int main(void)
{
    hello_from_gpu<<<4, 4>>>();
    hipDeviceSynchronize();

    return 0;

}